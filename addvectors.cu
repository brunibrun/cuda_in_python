#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
            exit(1);                                               \
        }                                                          \
    }


__global__ void add_vectors(double *a, double *b, double *out, const int num_rows)
{
    int idx;
    idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (; (idx < num_rows); idx += blockDim.x * gridDim.x)
        out[idx] = a[idx] + b[idx];
}


extern "C"
{
    void wrap_add_vectors(double *a, double *b, double *out, const int num_rows)
    {
        // alloc device memory
        size_t vec_bytes = num_rows * sizeof(double);

        double *a_device;
        double *b_device;
        double *out_device;

        CHECK(hipMalloc(&a_device, vec_bytes));
        CHECK(hipMalloc(&b_device, vec_bytes));
        CHECK(hipMalloc(&out_device, vec_bytes));

        // copy data to device
        CHECK(hipMemcpy(a_device, a, vec_bytes, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(b_device, b, vec_bytes, hipMemcpyHostToDevice));

        // launch kernel
        int threads_per_block = 256;
        int blocks_per_grid = (num_rows + threads_per_block - 1) / threads_per_block;

        add_vectors<<<threads_per_block, blocks_per_grid>>>(a_device, b_device, out_device, num_rows);

        CHECK(hipDeviceSynchronize());

        // copy data back to host
        CHECK(hipMemcpy(out, out_device, vec_bytes, hipMemcpyDeviceToHost));

        // free device memory
        CHECK(hipFree(a_device));
        CHECK(hipFree(b_device));
        CHECK(hipFree(out_device));

        CHECK(hipDeviceReset());
    }
}